#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>
#include <malloc.h>
#include"pgm_library.h"

// Funcion ejecutada en la GPU
__global__ void negativo(char *input_image, char *output_image, int nRows, int nCols){

	int r = blockDim.x*blockIdx.x + threadIdx.x;
	int i;

	// Each thread compute a row
	
	if (r < nRows){
	  for (i = 0; i < nCols; i++){
		  output_image[nCols*r + i] = 255 - input_image[nCols*r + i];
	  }
	}

}


// Main function
int main(int argc, char **argv){

  if (argc != 3){
    printf("\nUse: ./image_neg <input_file>.pgm <output_file>.pgm\n\n");
    exit(0);
  }


	image lena;

	// Reading image
	read_pgm(argv[1], &lena);

	char *pt = lena.data;
	int ndata = lena.rows*lena.cols;

/*
	//	Secuential programming
  int i;
	for (i = 0; i < lena.rows*lena.cols; i++){
		pt[i] = 255 - pt[i];
	}
*/

	// Parallel programming

	char *device_input_image = NULL;
	char *device_output_image = NULL;
	hipMalloc((void **) &device_input_image , ndata*sizeof(char));
	hipMalloc((void **) &device_output_image , ndata*sizeof(char));

	hipMemcpy(device_input_image, pt, ndata*sizeof(char), hipMemcpyHostToDevice);

	// Calling device function (using max 512 threads per block)
	int nBlocks = ceil(lena.rows/512.0);
	int nThreads = 512;
	
  negativo<<<nBlocks, nThreads>>>(device_input_image, device_output_image, lena.rows, lena.cols);

	hipMemcpy(pt, device_output_image, ndata*sizeof(char), hipMemcpyDeviceToHost);

	// Saving image
	write_pgm(argv[2], &lena);

	return 0;

}


